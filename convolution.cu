#include "hip/hip_runtime.h"
#include <iostream>
#include "convolution.h"

using namespace std;

const int filterSize = 33;

void *conv(ImageData num, double filter[4][33][33]){
    int num_height = sizeof(num.imgArray)/sizeof(num.imgArray[0]);
    int num_width = sizeof(num.imgArray[0])/sizeof(num.imgArray[0][0]);

    // float feature_maps[num_height-filterSize+1][num_width-filterSize+1][4];
    double (*result)[100][100] = new double[4][100][100];

    // double result[4][num_height-32][num_width-32];
    for(int i = 0; i < 4; i++){
        // Real convoluting happen here
        // changable for numpy.zeros
        for(int j = 0; j < num_height-32; j++){
            for(int k = 0; k < num_width-32; k++){
                result[i][j][k] = 0;
            }
            // cout << j << endl;
        }
        // disini tidak ditambah 1
        for(int r = filterSize/2.0; r < num_height-filterSize/2.0; r++){
            for(int c = filterSize/2.0; c < num_width-filterSize/2.0; c++){
                double curr_region[filterSize][filterSize];
                double sum = 0;
                int filterr = 0;
                for(int regionr = r-filterSize/2.0; regionr < r+filterSize/2.0-2; regionr++){
                    int filterc = 0;
                    for(int regionc = c-filterSize/2.0; regionc < c+filterSize/2.0-2; regionc++){
                        // cout << regionr << " " << regionc << " " << filterr << " " << filterc << endl;
                        // break;
                        curr_region[filterr][filterc] = num.imgArray[regionr][regionc] * filter[i][filterr][filterc];
                        sum += curr_region[filterr][filterc];
                        // cout << curr_region[regionr][regionc] << endl;
                        filterc++;
                    }
                    filterr++; 
                    // break;
                }
                result[i][r-16][c-16] = sum;
                // break;
            }
            // break;
        }
        // break;
    }
    return result;
    // return feature_maps;
}

void *pooling(){
    int stride = 2;
    int size = 2;
    int r2 = 0;
    for(int ro = 0; ro < height-size+1; ro+=stride){
        int c2 = 0;
        for(int co = 0; co < width-size+1; co+=stride){
            pool_out[r2,c2] = max(feature_maps[ro:ro+size, co:co+size]);
            
            c2++;
        }
    r2++;
    }
}
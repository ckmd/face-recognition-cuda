#include "hip/hip_runtime.h"
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/opencv.hpp>
#include <iostream>
#include <typeinfo>
#include <string>
#include <dirent.h>
#include "gabor.h"

using namespace std;
using namespace cv;

vector<int> getLabel(string name){
    vector<int> labelarray (22, 0);
    string label = name.substr(0, name.find(" "));
    if(label != "nonface"){
    //     cout << "face" << label << endl;
        labelarray[0] = 1;
        labelarray[stoi(label)] = 1;
    }
    return labelarray;
}

int main()
{
    struct ImageData { 
        int imgArray[100][100];
    } data;

    vector<String> filenames, filenamespng;
    vector<vector<int>> labels;
    vector<ImageData> datas;
    vector<int> label;

    // read image from file
    Mat img;
    String directory = "21pose/*.jpg";
    String directorypng = "21pose/*.png";
    string name;

    glob(directory, filenames);
    glob(directorypng, filenamespng);

    for(int a = 0; a < filenames.size() + filenamespng.size(); a++){
        if(a < filenames.size()){
            img = imread(filenames[a],1);
            if(!img.data){
                cout << "No image" << endl;
                return -1;
            }
            name = (string)filenames[a].substr(7,filenames[a].length()-1);
            label = getLabel(name);
        }else{
            img = imread(filenamespng[a-filenames.size()],1);
            if(!img.data){
                cout << "No image" << endl;
                return -1;
            }
            name = (string)filenamespng[a-filenames.size()].substr(7,filenamespng[a-filenames.size()].length()-1);
            label = getLabel(name);
        }
        cout << name;
        // Converting image from RGB into greyscale
        Mat grey;
        cvtColor( img, grey, CV_BGR2GRAY );
        
        // resizing image into 100x100
        Mat grey100;
        resize(grey, grey100, Size(100,100));
        imshow("test", grey100);
        waitKey(100);
        
        // converting Mat image into array
        for(int i = 0; i < grey100.rows; i++){
            for(int j = 0; j < grey100.cols; j++){
                data.imgArray[i][j] = (int)grey100.at<uchar>(i,j);
            }
        }
        labels.push_back(label);
        datas.push_back(data);
        cout << label[0] << " " << data.imgArray[0][0] << endl;
    }
    cout << labels.size() << " " << datas.size() << endl;
    // float (*map5)[maxMap][maxMap];
    // float min = load;
    // float map6 = map5[0][0][0];
    return 0;
}
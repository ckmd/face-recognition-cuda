#include "hip/hip_runtime.h"
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/opencv.hpp>
#include <iostream>
#include <typeinfo>
#include <string>
#include <dirent.h>
#include "gabor.h"

using namespace std;
using namespace cv;

vector<int> getLabel(string name){
    vector<int> labelarray (22, 0);
    string label = name.substr(0, name.find(" "));
    if(label != "nonface"){
    //     cout << "face" << label << endl;
        labelarray[0] = 1;
        labelarray[stoi(label)] = 1;
    }
    return labelarray;
}

int main()
{
    struct ImageData { 
        int imgArray[100][100];
    } data;

    vector<String> filenames;
    vector<vector<int>> labels;
    vector<ImageData> datas;

    // read image from file
    Mat img;
    String directory = "21pose/*.jpg";

    glob(directory, filenames);

    for(int a = 0; a < filenames.size(); a++){
        img = imread(filenames[a],1);
        if(!img.data){
            cout << "No image" << endl;
            return -1;
        }
        string name = (string)filenames[a].substr(7,filenames[a].length()-1);
        vector<int> label = getLabel(name);
        
        // Converting image from RGB into greyscale
        Mat grey;
        cvtColor( img, grey, CV_BGR2GRAY );
        
        // resizing image into 100x100
        Mat grey100;
        resize(grey, grey100, Size(100,100));
        
        // converting Mat image into array
        for(int i = 0; i < grey100.rows; i++){
            for(int j = 0; j < grey100.cols; j++){
                data.imgArray[i][j] = (int)grey100.at<uchar>(i,j);
            }
        }
        labels.push_back(label);
        datas.push_back(data);
        cout << labels.size() << " " << datas.size() << endl;
    }
    // float (*map5)[maxMap][maxMap];
    // float min = load;
    // float map6 = map5[0][0][0];
    return 0;
}
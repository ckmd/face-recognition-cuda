#include "hip/hip_runtime.h"
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/opencv.hpp>
#include <iostream>
#include <typeinfo>
#include <string>
#include <dirent.h>
#include "gabor.h"

using namespace std;
using namespace cv;

int * getLabel(string name){
    static int labelarray[22] = { };
    int label = stoi(name.substr(0, name.find(" ")));
    for(int i=0; i<22; i++){
        // Defining that is face
        labelarray[0] = 1;
        labelarray[label] = 1;
    }
    return labelarray;
}

int main()
{
    // read image from file
    Mat img;
    String directory = "21pose/*.jpg";
    vector<String> filenames;
    glob(directory, filenames);
    for(int a = 0; a < filenames.size(); a++){

        // string fileName = "1 person03175+30+45 Cropped.jpg";
        img = imread(filenames[a],1);
        if(!img.data){
            cout << "No image" << endl;
            return -1;
        }
        cout << filenames[a] << endl;
        
        // int *label = getLabel(fileName);
        // for(int i = 0; i < 22; i++){
        //     cout << *(label+i) << endl;
        // }
        
        // Converting image from RGB into greyscale
        Mat grey;
        cvtColor( img, grey, CV_BGR2GRAY );
        
        // resizing image into 100x100
        Mat grey100;
        resize(grey, grey100, Size(100,100));
        
        // converting Mat image into array
        int imgArray [100][100];
        for(int i = 0; i < grey100.rows; i++){
            for(int j = 0; j < grey100.cols; j++){
                imgArray[i][j] = (int)grey100.at<uchar>(i,j);
            }
        }
    }
    // float (*map5)[maxMap][maxMap];
    // float min = load;
    // float map6 = map5[0][0][0];
    for(int k=0; k < 4; k++){
        for (int i = 0; i < 9; i++){
            for(int j = 0; j < 9; j++)
            cout << map5i[k][i][j] << "\t";
            cout << endl;
        }
        cout << endl;
    }
    return 0;
}
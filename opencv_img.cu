#include "hip/hip_runtime.h"
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/opencv.hpp>
#include <iostream>
#include <typeinfo>
#include <string>
#include <dirent.h>
#include "gabor.h"
#include "convolution.h"

using namespace std;
using namespace cv;

vector<int> getLabel(string name){
    vector<int> labelarray (22, 0);
    string label = name.substr(0, name.find(" "));
    if(label != "nonface"){
    //     cout << "face" << label << endl;
        labelarray[0] = 1;
        labelarray[stoi(label)] = 1;
    }
    return labelarray;
}

double randZeroToOne()
{
    return rand() / (RAND_MAX + 1.);
}

struct ImageData { 
    int imgArray[132][132];
};

void conv(ImageData, float[4][maxMap][maxMap]);

int main()
{
    ImageData data;
    vector<String> filenames, filenamespng;
    vector<vector<int>> labels;
    vector<ImageData> datas;
    vector<int> label;

    // read image from file
    Mat img;
    String directory = "21pose/*.jpg";
    String directorypng = "21pose/*.png";
    string name;

    glob(directory, filenames);
    glob(directorypng, filenamespng);

    for(int a = 0; a < filenames.size() + filenamespng.size(); a++){
        if(a < filenames.size()){
            img = imread(filenames[a],1);
            if(!img.data){
                cout << "No image" << endl;
                return -1;
            }
            name = (string)filenames[a].substr(7,filenames[a].length()-1);
            label = getLabel(name);
        }else{
            img = imread(filenamespng[a-filenames.size()],1);
            if(!img.data){
                cout << "No image" << endl;
                return -1;
            }
            name = (string)filenamespng[a-filenames.size()].substr(7,filenamespng[a-filenames.size()].length()-1);
            label = getLabel(name);
        }
        // cout << name;
        // Converting image from RGB into greyscale
        Mat grey;
        cvtColor( img, grey, CV_BGR2GRAY );
        
        // resizing image into 100x100
        Mat grey100;
        resize(grey, grey100, Size(100,100));
        // imshow("test", grey100);
        // waitKey(1);
        
        // converting Mat image into array and add padding 16
        for(int i = 0; i < 132; i++){
            for(int j = 0; j < 132; j++){
                data.imgArray[i][j] = 128;
                if((i >= 16 && i < grey100.rows + 16) && (j >= 16 && j < grey100.cols + 16))
                    data.imgArray[i][j] = (int)grey100.at<uchar>(i,j);
            }
        }
        labels.push_back(label);
        datas.push_back(data);
        // cout << label[0] << " " << data.imgArray[0][0] << endl;
    }
    // cout << labels.size() << " " << datas.size() << endl;
    // float (*map5)[maxMap][maxMap];
    // float min = load;
    // float map6 = map5[0][0][0];

    // Neural Network Start Here
    double (*syn0)[1000] = new double[20000][1000];
    double (*syn1)[22] = new double[1000][22];
    double (*bias0)[1000] = new double[1][1000];
    double (*bias1)[22] = new double[1][22];

    for(int i = 0; i < 20000; i++)
        for(int j = 0; j < 1000; j++)
            syn0[i][j] = 2 * randZeroToOne() - 1;

    for(int i = 0; i < 1000; i++)
        for(int j = 0; j < 22; j++)
            syn1[i][j] = 2 * randZeroToOne() - 1;

    for(int i = 0; i < 1000; i++)
        bias0[0][i] = 2 * randZeroToOne() - 1;

    for(int i = 0; i < 22; i++)
        bias1[0][i] = 2 * randZeroToOne() - 1;
        
    double learning_rate = 0.005;

    // Training start here
    
    for(int i=0; i < 1; i++){
        (double (*)[maxMap][maxMap])conv(data, map33);
        // convolution
        // pooling
        // magnitude
        // phase
        // Forwardprop
        // BackProp
        // update weight
    }
    // save to pickle

    return 0;
}
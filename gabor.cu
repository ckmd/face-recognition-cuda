#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <math.h>
#include "gabor.h"

using namespace std;
// const int maxMap = 33;

double changeBaseX(double x, double y, double theta){
    double x_theta = x * cos(theta) + y * sin(theta);
    return x_theta;
}

double changeBaseY(double x, double y, double theta){
    double y_theta = y * cos(theta) - x * sin(theta);
    return y_theta;
}

double GaborFunction(double x, double y, double theta, double f, double sigma_x, double sigma_y){
    double r1 = changeBaseX(x,y,theta) / sigma_x;
    double r2 = changeBaseY(x,y,theta) / sigma_y;
    double arg = - 0.5 * ( pow(r1,2) + pow(r2,2) );
    return exp(arg) * cos(2*M_PI*f*changeBaseX(x,y,theta));
}

double GaborFunctionImajiner(double x, double y, double theta, double f, double sigma_x, double sigma_y){
    double r1 = changeBaseX(x,y,theta) / sigma_x;
    double r2 = changeBaseY(x,y,theta) / sigma_y;
    double arg = - 0.5 * ( pow(r1,2) + pow(r2,2) );
    return exp(arg) * sin(2*M_PI*f*changeBaseX(x,y,theta));
}

void *gabor(int radius, double freq, double sig_x, double sig_y){
    int area = radius*2+1;
    double (*M)[maxMap][maxMap] = new double[4][maxMap][maxMap];
    int id = 0;
    for(double theta = 0.0; theta < 180.0; theta += 45.0){
        double x = -(double)radius;
//        cout << theta << endl;
        for(int i = 0; i < area; i++){
            double y = -(double)radius;
            for(int j = 0; j <area; j++){
                M[id][i][j] = 0;
                M[id][i][j] = GaborFunction(x,y,theta*M_PI/180,freq,sig_x,sig_y);
                y = y + 1;
//                cout << M[id][i][j] << "\t";
            }
//            cout << endl;
            x = x + 1;
        }
    id = id + 1;
//    cout << endl;
    }
    return M;
}

void *gaborImajiner(int radius, double freq, double sig_x, double sig_y){
    int area = radius*2+1;
    double (*M)[maxMap][maxMap] = new double[4][maxMap][maxMap];
    int id = 0;
    for(double theta = 0.0; theta < 180.0; theta += 45.0){
        double x = -(double)radius;
//        cout << theta << endl;
        for(int i = 0; i < area; i++){
            double y = -(double)radius;
            for(int j = 0; j <area; j++){
                M[id][i][j] = 0;
                M[id][i][j] = GaborFunctionImajiner(x,y,theta*M_PI/180,freq,sig_x,sig_y);
                y = y + 1;
//                cout << M[id][i][j] << "\t";
            }
//            cout << endl;
            x = x + 1;
        }
    id = id + 1;
//    cout << endl;
    }
    return M;
}


double (*map5)[maxMap][maxMap] = ( double (*)[maxMap][maxMap])gabor(2, 0.6, 1.25, 1.25);
double (*map9)[maxMap][maxMap] = ( double (*)[maxMap][maxMap]) gabor(4, 0.44, 1.45, 1.45);
double (*map17)[maxMap][maxMap] = ( double (*)[maxMap][maxMap]) gabor(8, 0.22, 2.7, 2.7);
double (*map33)[maxMap][maxMap] = ( double (*)[maxMap][maxMap]) gabor(16, 0.12, 5.1, 5.1);

double (*map5i)[maxMap][maxMap] = ( double (*)[maxMap][maxMap]) gaborImajiner(2, 0.6, 1.25, 1.25);
double (*map9i)[maxMap][maxMap] = ( double (*)[maxMap][maxMap]) gaborImajiner(4, 0.44, 1.45, 1.45);
double (*map17i)[maxMap][maxMap] = ( double (*)[maxMap][maxMap]) gaborImajiner(8, 0.22, 2.7, 2.7);
double (*map33i)[maxMap][maxMap] = ( double (*)[maxMap][maxMap]) gaborImajiner(16, 0.12, 5.1, 5.1);
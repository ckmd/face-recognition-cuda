#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <math.h>
#include "gabor.h"

using namespace std;
// const int maxMap = 33;

float changeBaseX(float x, float y, float theta){
    float x_theta = x * cos(theta) + y * sin(theta);
    return x_theta;
}

float changeBaseY(float x, float y, float theta){
    float y_theta = y * cos(theta) - x * sin(theta);
    return y_theta;
}

float GaborFunction(float x, float y, float theta, float f, float sigma_x, float sigma_y){
    float r1 = changeBaseX(x,y,theta) / sigma_x;
    float r2 = changeBaseY(x,y,theta) / sigma_y;
    float arg = - 0.5 * ( pow(r1,2) + pow(r2,2) );
    return exp(arg) * cos(2*M_PI*f*changeBaseX(x,y,theta));
}

float GaborFunctionImajiner(float x, float y, float theta, float f, float sigma_x, float sigma_y){
    float r1 = changeBaseX(x,y,theta) / sigma_x;
    float r2 = changeBaseY(x,y,theta) / sigma_y;
    float arg = - 0.5 * ( pow(r1,2) + pow(r2,2) );
    return exp(arg) * sin(2*M_PI*f*changeBaseX(x,y,theta));
}

void *gabor(int radius, float freq, float sig_x, float sig_y){
    int area = radius*2+1;
    float (*M)[maxMap][maxMap] = new float[4][maxMap][maxMap];
    int id = 0;
    for(float theta = 0.0; theta < 180.0; theta += 45.0){
        float x = -(float)radius;
//        cout << theta << endl;
        for(int i = 0; i < area; i++){
            float y = -(float)radius;
            for(int j = 0; j <area; j++){
                M[id][i][j] = 0;
                M[id][i][j] = GaborFunction(x,y,theta*M_PI/180,freq,sig_x,sig_y);
                y = y + 1;
//                cout << M[id][i][j] << "\t";
            }
//            cout << endl;
            x = x + 1;
        }
    id = id + 1;
//    cout << endl;
    }
    return M;
}

void *gaborImajiner(int radius, float freq, float sig_x, float sig_y){
    int area = radius*2+1;
    float (*M)[maxMap][maxMap] = new float[4][maxMap][maxMap];
    int id = 0;
    for(float theta = 0.0; theta < 180.0; theta += 45.0){
        float x = -(float)radius;
//        cout << theta << endl;
        for(int i = 0; i < area; i++){
            float y = -(float)radius;
            for(int j = 0; j <area; j++){
                M[id][i][j] = 0;
                M[id][i][j] = GaborFunctionImajiner(x,y,theta*M_PI/180,freq,sig_x,sig_y);
                y = y + 1;
//                cout << M[id][i][j] << "\t";
            }
//            cout << endl;
            x = x + 1;
        }
    id = id + 1;
//    cout << endl;
    }
    return M;
}


float (*map5)[maxMap][maxMap] = ( float (*)[maxMap][maxMap])gabor(2, 0.6, 1.25, 1.25);
float (*map9)[maxMap][maxMap] = ( float (*)[maxMap][maxMap]) gabor(4, 0.44, 1.45, 1.45);
float (*map17)[maxMap][maxMap] = ( float (*)[maxMap][maxMap]) gabor(8, 0.22, 2.7, 2.7);
float (*map33)[maxMap][maxMap] = ( float (*)[maxMap][maxMap]) gabor(16, 0.12, 5.1, 5.1);

float (*map5i)[maxMap][maxMap] = ( float (*)[maxMap][maxMap]) gaborImajiner(2, 0.6, 1.25, 1.25);
float (*map9i)[maxMap][maxMap] = ( float (*)[maxMap][maxMap]) gaborImajiner(4, 0.44, 1.45, 1.45);
float (*map17i)[maxMap][maxMap] = ( float (*)[maxMap][maxMap]) gaborImajiner(8, 0.22, 2.7, 2.7);
float (*map33i)[maxMap][maxMap] = ( float (*)[maxMap][maxMap]) gaborImajiner(16, 0.12, 5.1, 5.1);
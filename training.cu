
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <math.h>
#define epoch 100000000
using namespace std;

int rand();
double RandomNumber(double Min, double Max)
{
    return ((double(rand()) / double(RAND_MAX)) * (Max - Min)) + Min;
}


double sigmoid(double x){
    return 1 / (1 + exp(-x));
}

double sigmoid_der(double x){
    return sigmoid(x) * (1 - sigmoid(x));
}

float * dot_matrix(float m1[3], float m2[3]){
    static float C[1];

    C[0] = 0;
    for (int j = 0; j < 3; j++){
        C[0] +=  m1[j] * m2[j];
    }
    return C;
}

float weight[3];
float bias = RandomNumber(-1,1);
float learning_rate = 0.05;
float *inputs, suminput, activation1;
float error, dcost_dpred, dpred_dz, z_delta;
int ri;
double sum_error = 0;

void testing(float in[]){
    inputs = dot_matrix(in, weight);
    suminput = inputs[0] + bias;
    activation1 = sigmoid(suminput);
    cout << activation1 << endl;
}

int main(){

    cout << "Neural Network Start" << endl;
    float feature_set[5][3] = {{0,1,0},{0,0,1},{1,0,0},{1,1,0},{1,1,1}};
    float label[5][1] = {{1},{0},{0},{1},{1}};

    // filling weight with random number
    for(int i = 0; i < 3; i++){
        weight[i] = RandomNumber(-1, 1);
    }
    // Training Phase
    cout << "Training Section" << endl;
    for(int i = 0; i < epoch; i++){
        ri = rand() % 5;

        inputs = dot_matrix(feature_set[ri], weight);
        suminput = inputs[0] + bias;
        activation1 = sigmoid(suminput);

        error = activation1 - label[ri][0];

        dcost_dpred = error;
        dpred_dz = sigmoid_der(activation1);
        z_delta = dcost_dpred * dpred_dz;

        for(int j = 0; j < 3; j++){
            weight[j] -= (learning_rate * feature_set[ri][j] * z_delta);
        }
        bias -= learning_rate * z_delta;
        sum_error += error;

        if(i % 10000000 == 0 && i > 0){
            cout << "Epoch " << i << " error : " << sum_error / (i+1) << endl;
        }

    }
    cout << "Testing Section" << endl;
    for(int i = 0; i < 5; i++)
        testing(feature_set[i]);

    return 0;
}
